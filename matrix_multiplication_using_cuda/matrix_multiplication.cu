#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

#define N 1024
// We use it for a matrix of dimension N x N

// Kernel for parallel matrix multiplication
__global__ void matMulParallel(float *A, float *B, float *C, int start, int end){
    /*
    Note: A kernal function is a function that is executed on the GPU...
    The __global__ keyword in CUDA marks this function as a kernel, 
    meaning it can be called from the host (CPU) and executed on the device (GPU)... 
    */
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    /*
    blockIdx, blockDim, and threadIdx are special variables in CUDA that provide the index of the block and thread within that block...
    blockIdx gives the index of the block in the grid.
    blockDim gives the number of threads per block.
    threadIdx gives the index of the thread within the block.
    */
    if(row>=start && row< end && col<N){
        float value = 0;
        for (int k=0; k<N; k++){
            value+=A[row*N+k]*B[k*N+col];
        }
        C[row*N+col]=value;
    }
}

// sequential matrix multiplication
void matMulSequential(float *A, float *B, float *C, int start, int end){
    for (int i = start; i < end; i++) {
        for (int j = 0; j < N; j++) {
            float value = 0;
            for (int k = 0; k < N; k++) {
                value += A[i * N + k] * B[k * N + j];
            }
            C[i * N + j] = value;
        }
    }
}

int main(int argc, char **argv){
    if(argc!=2){
        cerr << "Usage: "<< argv[0] << " <sequential_percentage>" << endl;
        return -1;
    }

    float sequentialPercentage = atof(argv[1]);

    if(sequentialPercentage<0.0f || sequentialPercentage>1.0f){
        // the break cases
        cerr << "Error: sequential percentage must be between 0.0 and 1.0" << endl;
        return -1;
    }

    // host matrices -  the matrices accessible by the cpu
    float *h_A, *h_B, *h_C;
    // device matrices - the matrices accessible by the gpu
    float *d_A, *d_B, *d_C;

    // note that we are considering the matrices as a 1 dimentional array here

    size_t bytes = N*N*sizeof(float);

    // allocate host memory
    h_A = (float *)malloc(bytes);
    h_B = (float *)malloc(bytes);
    h_C = (float *)malloc(bytes);

    // initializing matrices
    // note that i've initialised matrices in a basic manner it can be made better (like randomizing it) if need be
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
    }

    // allocate device memory
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // copying data from host to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // now definig the sequential and parallel split
    int sequentialRows = N*sequentialPercentage;
    int parallelRows = N - sequentialRows;

    // measuring the execution time
    auto start = chrono::high_resolution_clock::now();

    // sequential computation
    matMulSequential(h_A, h_B, h_C, 0, sequentialRows);


    // parallel computation
    dim3 threads(16,16);
    dim3 blocks((N + threads.x-1)/threads.x, (N+threads.y-1)/threads.y);
    matMulParallel<<<blocks, threads>>>(d_A, d_B, d_C, sequentialRows, N);
    // the above line launches the cuda kernel with the specified grid and block dimenstions
    
    hipDeviceSynchronize();
    hipMemcpy(h_C+sequentialRows*N, d_C+sequentialRows*N, parallelRows*N*sizeof(float), hipMemcpyDeviceToHost);
    // copies the memory in the fromat (destination, source, size, direction)
    auto end = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed = end - start;

    cout << "Execution time: " << elapsed.count() << " seconds" << endl;

    // freedom #erenYaegar
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;

}